#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixMul(float *A, float *B, float *C, int N) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; k++) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

int main() {
    const int N = 5;
    size_t size = N * N * sizeof(float);
    float *A, *B, *C, *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize 5x5 matrices with some sample values
    for (int i = 0; i < N * N; i++) {
        A[i] = (i % N) + 1;        // e.g., 1, 2, 3, 4, 5, 1, 2, ...
        B[i] = ((i % N) + 1) * 2;  // e.g., 2, 4, 6, 8, 10, 2, 4, ...
    }

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy input matrices to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define thread hierarchy
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize(); // Ensure kernel finishes

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Display matrices and result
    std::cout << "Matrix A:\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << A[i * N + j] << "\t";
        std::cout << "\n";
    }

    std::cout << "\nMatrix B:\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << B[i * N + j] << "\t";
        std::cout << "\n";
    }

    std::cout << "\nResult Matrix C (A x B):\n";
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << C[i * N + j] << "\t";
        std::cout << "\n";
    }

    // Clean up
    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    return 0;
}
